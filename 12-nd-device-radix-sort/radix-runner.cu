#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <tuple>

#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "radix.cu"

template<typename T>
void createData(uint32_t size, T* d_sort, uint32_t* d_idx, T* h_sort, uint32_t* h_idx, bool seq) {
    uint32_t sortsize = size * sizeof(T);
    uint32_t idxsize  = size * sizeof(uint32_t);


    for(uint32_t i=0; i<size; i++) {
        if(seq) {
            h_sort[i] = static_cast<T>(i);
        } else {
            // TODO
        }
        h_idx[i] = i;
    }

    hipMemcpy(d_sort, h_sort, sortsize, hipMemcpyHostToDevice);
    hipMemcpy(d_idx, h_idx, idxsize, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
}

// Helper functions for bit conversions
template<typename T>
inline uint32_t toBitsCpu(T val) {
    if constexpr (std::is_same<T, float>::value) {
        uint32_t fuint;
        memcpy(&fuint, &val, sizeof(float));
        return fuint ^ ((fuint >> 31) | 0x80000000);
    }
    else if constexpr (std::is_same<T, __half>::value) {
        uint16_t bits = __half_as_ushort(val);
        uint16_t mask = -int(bits >> 15) | 0x8000;
        return static_cast<uint32_t>(bits ^ mask);
    }
    else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
        uint16_t bits = __bfloat16_as_ushort(val);
        uint16_t mask = -int(bits >> 15) | 0x8000;
        return static_cast<uint32_t>(bits ^ mask);
    }
    else if constexpr (std::is_same<T, int64_t>::value) {
        // return static_cast<uint32_t>((val >> radixShift) & 0xFFFFFFFF);
        // TODO - how to handle int64?????
    }
    else {
        return static_cast<uint32_t>(val);
    }
}

// Calculate resources to run
struct Resources {
    uint32_t numElemInBlock; // Elements per block
    uint32_t numVecElemInBlock; // Vector elements per block
    uint32_t numThreadBlocks; // number of threadblocks to run for Upsweep and DownsweepPairs kernel
    uint32_t const numUpsweepThreads = 256; // Num threads per upsweep kernel

    uint32_t const radix = RADIX;

    static Resources compute(uint32_t size, uint32_t type_size) {
        Resources res;

        // Query device properties
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        
        // Calculate shared memory needed for per-block histogram
        // This corresponds to __shared__ uint32_t s_globalHist[RADIX * 2] in the kernel
        const uint32_t shared_hist_size = res.radix * 2 * sizeof(uint32_t);

        // Calculate available shared memory for data processing
        const uint32_t available_shared_mem = (prop.sharedMemPerBlock - shared_hist_size) * 3 / 4;  // Use ~75% of remaining shared memory

        // Calculate part_size based on shared memory constraints
        res.numElemInBlock = available_shared_mem / type_size;
        // For 4-byte types, adjust vec_part_size for vector loads
        res.numVecElemInBlock = (type_size == 4) ? 
            res.numElemInBlock / 4 : res.numElemInBlock;

        res.numThreadBlocks = (size + res.numElemInBlock - 1) / res.numElemInBlock;
        return res;
    }
};

template<typename T>
uint32_t validateUpsweep(uint32_t size, bool dataseq = true) {
    printf("Validating upsweep for size[%u] and typeSize[%lu]\n", size, sizeof(T));
    uint32_t errors = 0;

    Resources res = Resources::compute(size, sizeof(uint32_t));
    printf("For size[%u] -------------\nnumThreadBlocks: %u numUpsweepThreads: %u numElementsInBlock: %u numVecElementsInBlock: %u\n", size, res.numThreadBlocks, res.numUpsweepThreads, res.numElemInBlock, res.numVecElemInBlock);
    
    // Declarations
    T* d_sort;
    T* d_sortAlt;
    uint32_t* d_idx;
    
    uint32_t* d_globalHist;
    uint32_t* d_passHist;

    uint32_t numPasses = sizeof(T);
    uint32_t sortSize  = size * sizeof(T);
    uint32_t idxSize   = size * sizeof(uint32_t);
    uint32_t radixSize = RADIX * sizeof(uint32_t);

    T* h_sort       = (T*)malloc(sortSize);
    uint32_t* h_idx = (uint32_t*)malloc(idxSize);

    hipMalloc(&d_sort, sortSize);
    hipMalloc(&d_sortAlt, sortSize);
    hipMalloc(&d_idx, idxSize);
    // hipMalloc(&d_sortAlt, sortSize);
    hipMalloc(&d_globalHist, radixSize * numPasses);
    hipMalloc(&d_passHist, radixSize * res.numThreadBlocks);

    // Create some data
    createData<T>(size, d_sort, d_idx, h_sort, h_idx, dataseq);

    hipMemset(d_globalHist, 0,  radixSize * numPasses);
    hipDeviceSynchronize();

    for(uint32_t pass=0; pass < numPasses; pass++) {
        uint32_t shift = pass * 8;
        RadixUpsweep<T><<<res.numThreadBlocks, res.numUpsweepThreads>>>(d_sort, d_globalHist, d_passHist, size, shift, res.numElemInBlock, res.numVecElemInBlock);

        uint32_t *cpuHist = (uint32_t*)malloc(radixSize);
        uint32_t *gpuHist = (uint32_t*)malloc(radixSize);

        for(int i=0; i<RADIX; i++) {
            cpuHist[i] = 0;
        }

        // Compute CPU histogram
        for (uint32_t i = 0; i < size; i++) {
            uint32_t bits = toBitsCpu<T>(h_sort[i]);
            uint32_t digit = (bits >> shift) & RADIX_MASK;
            cpuHist[digit]++;
        }
        // Convert to exclusive prefix sum
        uint32_t prev = 0;
        for (uint32_t i = 0; i < RADIX; i++) {
            uint32_t current = cpuHist[i];
            cpuHist[i] = prev;
            prev += current;
        }

        hipMemcpy(gpuHist, d_globalHist + (RADIX * pass), radixSize, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        
        for(uint32_t i=0; i<RADIX; i++) {
            if(cpuHist[i] != gpuHist[i]) {
                errors += 1;
                printf("Error[bin %u/ radixShift %u]: CPU[%u] GPU[%u]\n", i, shift, cpuHist[i], gpuHist[i]);
            }
        }
        
        free(cpuHist);
        free(gpuHist);
    }

    hipFree(d_sort);
    hipFree(d_sortAlt);
    hipFree(d_idx);
    hipFree(d_globalHist);
    hipFree(d_passHist);

    free(h_sort);
    free(h_idx);
    return errors;
}

int main() {
    uint32_t sizes[] = { 16, 1024, 2048, 4096, 4113, 7680, 8192, 32000, 64000, 128000 };
    
    // First, test for UpsweepKernel is good?
    for(uint32_t i = 0; i < 10; i++) {
        {
            printf("`uint32_t`: Upsweep Validation\n");
            uint32_t errors = validateUpsweep<uint32_t>(sizes[i]);
            if(errors > 0)
                printf("Errors: %u while validating upsweep for size[uint32_t][%u]\n", errors, sizes[i]);
        }

        {
            printf("`float`: Upsweep Validation\n");
            uint32_t errors = validateUpsweep<float>(sizes[i]);
            if(errors > 0)
                printf("Errors: %u while validating upsweep for size[float][%u]\n", errors, sizes[i]);
        }

        // {
        //     uint32_t errors = validateUpsweep<half>(sizes[i]);
        //     if(errors > 0)
        //         printf("Errors: %u while validating upsweep for size[float16][%u]", errors, sizes[i]);
        // }
    }
    return 0;
}