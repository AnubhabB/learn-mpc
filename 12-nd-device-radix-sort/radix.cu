
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#define N_THREADS           256
#define RADIX               256             //Number of digit bins
#define WARP_SIZE           32
#define LANE_LOG            5               // LANE_LOG = 5 since 2^5 = 32 = warp size
#define RADIX_LOG           8               // 2^8 = 258

#define LANE_MASK           (WARP_SIZE - 1)
#define RADIX_MASK          (RADIX - 1)     //Mask of digit bins, to extract digits
#define WARP_INDEX          (threadIdx.x >> LANE_LOG)

#define BIN_KEYS_PER_THREAD 15
#define SUB_PARTITION_SIZE (BIN_KEYS_PER_THREAD * WARP_SIZE);

// Thread position within a warp
__device__ __forceinline__ uint32_t getLaneId() {
    uint32_t laneId;
    asm("mov.u32 %0, %%laneid;" : "=r"(laneId));
    return laneId;
}

// Gets a bit mask representing all lanes with IDs less than the current thread's lane ID within a warp
__device__ __forceinline__ unsigned getLaneMaskLt() {
    unsigned mask;
    asm("mov.u32 %0, %%lanemask_lt;" : "=r"(mask));
    return mask;
}

// Prefix sum
// Performs an inclusive scan operation within a single warp
// Uses butterfly/sequential addressing pattern for efficiency
__device__ __forceinline__ uint32_t InclusiveWarpScan(uint32_t val) {
    #pragma unroll
    for (int offset = 1; offset <= 16; offset <<= 1) {
        const uint32_t t = __shfl_up_sync(0xffffffff, val, offset);
        if (getLaneId() >= offset) val += t;
    }

    return val;
}

// Circular shift prefix sum
__device__ __forceinline__ uint32_t InclusiveWarpScanCircularShift(uint32_t val) {
    #pragma unroll
    for (int offset = 1; offset <= 16; offset <<= 1) {
        const uint32_t t = __shfl_up_sync(0xffffffff, val, offset);
        if (getLaneId() >= offset) val += t;
    }

    return __shfl_sync(0xffffffff, val, getLaneId() + LANE_MASK & LANE_MASK);
}

// Helper for active warp scan (used for inter-warp scan) with early termination
__device__ __forceinline__ uint32_t ActiveInclusiveWarpScan(uint32_t val) {
    const uint32_t mask = __activemask();
    const int active_threads = __popc(mask);

    #pragma unroll
    for (int offset = 1; offset <= 16; offset <<= 1) {
        if (offset >= active_threads) break;  // Early termination
        const uint32_t t = __shfl_up_sync(mask, val, offset);
        if (getLaneId() >= offset) val += t;
    }

    return val;
}

// Prefix sum of active threads ot excluding itself
__device__ __forceinline__ uint32_t ActiveExclusiveWarpScan(uint32_t val) {
    const uint32_t mask = __activemask();
    #pragma unroll
    for (int offset = 1; offset <= 16; offset <<= 1) {
        const uint32_t t = __shfl_up_sync(mask, val, offset);
        if (getLaneId() >= offset) val += t;
    }

    const uint32_t t = __shfl_up_sync(mask, val, 1);
    return getLaneId() ? t : 0;
}

// Helper functions for bit conversions
template<typename T>
__device__ inline uint32_t toBits(T val) {
    if constexpr (std::is_same<T, float>::value) {
        return __float_as_uint(val) ^ ((__float_as_uint(val) >> 31) | 0x80000000);
    }
    else if constexpr (std::is_same<T, __half>::value) {
        uint16_t bits = __half_as_ushort(val);
        uint16_t mask = -int(bits >> 15) | 0x8000;
        return static_cast<uint32_t>(bits ^ mask);
    }
    else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
        uint16_t bits = __bfloat16_as_ushort(val);
        uint16_t mask = -int(bits >> 15) | 0x8000;
        return static_cast<uint32_t>(bits ^ mask);
    }
    else if constexpr (std::is_same<T, int64_t>::value) {
        // return static_cast<uint32_t>((val >> radixShift) & 0xFFFFFFFF);
        // TODO - how to handle int64?????
    }
    else {
        return static_cast<uint32_t>(val);
    }
}

// Vector type mappings
template<typename T> struct VectorTrait {
    // Default case (32-bit types)
    static constexpr uint32_t vector_size = 4;  // 4 * 4 bytes = 16 bytes
    static constexpr uint32_t bytes_per_vector = sizeof(T) * vector_size;
};

// Vectorizations for different types
template<>
struct VectorTrait<half> {
    static constexpr uint32_t vector_size = 8;  // 8 * 2 bytes = 16 bytes
    // using type = float4;
};

template<>
struct VectorTrait<hip_bfloat16> {
    static constexpr uint32_t vector_size = 8;   // 8 * 2 bytes = 16 bytes
};

// Specialization for 8-bit types
template<>
struct VectorTrait<uint8_t> {
    static constexpr uint32_t vector_size = 16;  // 16 * 1 byte = 16 bytes
};


// Helper function to get type-specific maximum value
template<typename T>
__device__ inline T getTypeMax() {
    if constexpr (std::is_same<T, float>::value) {
        return INFINITY;
    }
    else if constexpr (std::is_same<T, __half>::value) {
        return __float2half(INFINITY);
    }
    else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
        return __float2bfloat16(INFINITY);
    }
    else if constexpr (std::is_same<T, int64_t>::value) {
        return 0x7FFFFFFFFFFFFFFF;
    } else if constexpr (std::is_same<T, unsigned char>::value) {
        return 0xFF; // 255 in hex
    } else if constexpr (std::is_same<T, u_int32_t>::value) {
        return 0xFFFFFFFF;  // 4294967295 in hex
    } else {
        // This seems to be experimental
        // calling a constexpr __host__ function("max") from a __device__ function("getTypeMax") is not allowed. The experimental flag '--expt-relaxed-constexpr' can be used to allow this.
        
        // Shouldn't reach here
        return static_cast<T>(-1);
    }
}

// Radix Upsweep pass does the following:
// radixShift - signifies which `digit` position is being worked on in strides of 8 - first pass for MSB -> last 8 bits using radix 256
// passHist - for a particular digit position creates a frequency of values -
// in this implementaiton a passHist is computer per threadBlock and each threadBlock is responsible for processing `numElementsInBlock`
// globalHist - converts these frequencies into cumulative counts (prefix sums)
template<typename T>
__global__ void RadixUpsweep(
    T* sort,
    uint32_t* globalHist,
    uint32_t* passHist,
    const uint32_t size,
    const uint32_t radixShift,
    const uint32_t maxElemInBlock // max number of elements being processed by this block
) {
    uint32_t printBlock = 0;
    // Shared memory for histogram - two sections to avoid bank conflicts
    constexpr uint32_t sharedSize = RADIX * 2;
    __shared__ uint32_t s_globalHist[sharedSize];

    // Clear shared memory histogram
    #pragma unroll
    for (uint32_t i = threadIdx.x; i < sharedSize; i += blockDim.x)
        s_globalHist[i] = 0;
    __syncthreads();

    // Calculate this block's range
    const uint32_t block_start = blockIdx.x * maxElemInBlock;
    const uint32_t block_end = min(block_start + maxElemInBlock, size);
    const uint32_t elements_in_block = block_end - block_start;

    // if(blockIdx.x == printBlock && (threadIdx.x == 0 || threadIdx.x == 15)) {
    //     printf("Thread[%u]: maxElemInBlock[%u] block_start[%u] block_end[%u] elements_in_block[%u]\n", threadIdx.x, maxElemInBlock, block_start, block_end, elements_in_block);
    // }

    // if(threadIdx.x == 0 && blockIdx.x == printBlock) {
    //     // printf("\nBlock[%u]: ", blockIdx.x);
    //     for(uint32_t i=block_start; i<block_end; ++i) {
    //         printf("%u ", sort[i]);
    //     }
    //     printf("--\n");
    // }
    // Vector load based on types
    constexpr uint32_t vec_size = VectorTrait<T>::vector_size;

    // Calculate number of full vectors - we are going to make an attempt to process
    const uint32_t full_vecs = elements_in_block / vec_size;
    const uint32_t vec_end = block_start + (full_vecs * vec_size);
    
    for (uint32_t i = threadIdx.x; i < full_vecs; i += blockDim.x) {
        const uint32_t idx = block_start + i * vec_size;
        
        if(idx < vec_end) {
            // if(blockIdx.x == printBlock) {
            //     printf("Idx[%u] ", idx);
            // }
            #pragma unroll
            for (int j = 0; j < vec_size; ++j) {
                uint32_t bits = toBits(sort[idx + j]);
                // if(blockIdx.x == printBlock) {
                //     printf("sort[%u %u]: [%u %u %u] ", idx, idx + j, sort[idx + j], bits, bits >> radixShift & RADIX_MASK);
                // }
                atomicAdd(&s_globalHist[bits >> radixShift & RADIX_MASK], 1);
            }
        }
    }
    
    // Process remaining elements
    for (uint32_t i = threadIdx.x + vec_end; i < block_end; i += blockDim.x) {
        uint32_t bits = toBits(sort[i]);
        atomicAdd(&s_globalHist[bits >> radixShift & RADIX_MASK], 1);
    }

    __syncthreads();

    // Reduce histograms and prepare for prefix sum
    for (uint32_t i = threadIdx.x; i < RADIX; i += blockDim.x) {
        // Merge possible bank conflicts
        s_globalHist[i] += s_globalHist[i + RADIX];
        // Memory layout: digit frequencies across all blocks
        // So, if we have n blocks we'll have frequency values for a digit in each blocks consecutively
        passHist[i * gridDim.x + blockIdx.x] = s_globalHist[i];
        s_globalHist[i] = InclusiveWarpScanCircularShift(s_globalHist[i]);
    }
    __syncthreads();
    // if(threadIdx.x == 15 && blockIdx.x == printBlock) {
    //     printf("\nBlockHist[%u]: ", blockIdx.x);
    //     for(uint32_t i=0; i<RADIX; ++i) {
    //         printf("[%u %u %u]\n", i, passHist[RADIX * blockIdx.x + i], s_globalHist[i]);
    //     }
    //     printf("--\n");
    // }

    // Perform warp-level scan - for first thread in each warp
    if (threadIdx.x < (RADIX >> LANE_LOG))
        s_globalHist[threadIdx.x << LANE_LOG] = ActiveExclusiveWarpScan(s_globalHist[threadIdx.x << LANE_LOG]);
    __syncthreads();

    // if(threadIdx.x == 15 && blockIdx.x == printBlock) {
    //     printf("\nBlockHist[%u]: ", blockIdx.x);
    //     for(uint32_t i=0; i<RADIX; ++i) {
    //         printf("[%u %u %u]\n", i, passHist[RADIX * blockIdx.x + i], s_globalHist[i]);
    //     }
    //     printf("--\n");
    // }
    // Update global histogram with prefix sum results
    for (uint32_t i = threadIdx.x; i < RADIX; i += blockDim.x) {
        atomicAdd(
            &globalHist[i + (radixShift << LANE_LOG)], 
            s_globalHist[i] + 
            (getLaneId() ? 
                __shfl_sync(0xfffffffe, s_globalHist[i - 1], 1) : 0)
        );
    }

    // if(threadIdx.x == 15 && blockIdx.x == printBlock) {
    //     printf("\nBlockHist[%u]: ", blockIdx.x);
    //     for(uint32_t i=0; i<RADIX; ++i) {
    //         printf("[%u %u %u]\n", i, passHist[RADIX * blockIdx.x + i], s_globalHist[i]);
    //     }
    //     printf("--\n");
    // }
}


__global__ void RadixScan(
    uint32_t* passHist,
    const uint32_t numBlocks
) {
    const uint32_t blockSize = blockDim.x;
    const uint32_t tid = threadIdx.x;
    const uint32_t laneId = getLaneId();

    __shared__ uint32_t s_scan[128];

    // Initialize the shared memory!
    s_scan[tid] = 0;
    __syncthreads();

    
    // Circular shift within warp - this helps reduce bank conflicts
    // Get ID of the next thread: getLaneId(): 0 -> 1, 1 -> 2 ... 31 -> 0
    const uint32_t circularLaneShift = (laneId + 1) & LANE_MASK;
    
    if(threadIdx.x == 24 && blockIdx.x == 4) {
        printf("Lane: %u %u\n", circularLaneShift, laneId);
    }

    // Where does the digit start
    const uint32_t digitOffset = blockIdx.x * numBlocks;

    // Calculate the number of full block-sized chunks we need to process
    const uint32_t fullBlocksEnd = (numBlocks / blockSize) * blockSize;
    
    // Running sum for carrying over between iterations
    uint32_t reduction = 0;

    // Process full blocks
    // for (uint32_t blockStart = 0; blockStart < fullBlocksEnd; blockStart += blockSize) {
    //     // Load data into shared memory with circular shift pattern
    //     const uint32_t globalIdx = blockStart + tid;
    //     s_scan[tid] = passHist[globalIdx + digitOffset];

    //     // Step 1: Perform warp-level scan
    //     s_scan[tid] = InclusiveWarpScan(s_scan[tid]);
    //     __syncthreads();

    //     // Step 2: Collect and scan warp totals
    //     if (tid < (blockDim.x >> LANE_LOG)) {
    //         s_scan[((tid + 1) << LANE_LOG) - 1] = ActiveInclusiveWarpScan(s_scan[((tid + 1) << LANE_LOG) - 1]);
    //     }
    //     __syncthreads();

    //     const uint32_t writeIdx = circularLaneShift + (blockStart & ~LANE_MASK);
    //     if (writeIdx < numBlocks) {
    //         passHist[writeIdx + digitOffset] =
    //             (laneId != LANE_MASK ? s_scan[tid] : 0) +
    //             (tid >= WARP_SIZE ? 
    //                 __shfl_sync(0xffffffff, s_scan[tid - 1], 0) : 0) +
    //             reduction;
    //     }

    //     reduction += s_scan[blockSize - 1];
    //     __syncthreads();
    // }
    uint32_t tidx = tid;
    for(; tidx<fullBlocksEnd; tidx += blockDim.x) {
        s_scan[tid] = passHist[tid + digitOffset];
        
        // Perform warp-level scan
        s_scan[tid] = InclusiveWarpScan(s_scan[tid]);
        __syncthreads();

        // Collect and scan warp totals
        if (tid < (blockDim.x >> LANE_LOG)) {
            s_scan[((tid + 1) << LANE_LOG) - 1] = ActiveInclusiveWarpScan(s_scan[((tid + 1) << LANE_LOG) - 1]);
        }
        __syncthreads();

        const uint32_t writeIdx = circularLaneShift + (tidx & ~LANE_MASK);

        passHist[writeIdx + digitOffset] =
            (getLaneId() != LANE_MASK ? s_scan[tid] : 0) +
            (tid >= WARP_SIZE ?
            s_scan[(tid & ~LANE_MASK) - 1] : 0) +
            reduction;

        reduction += s_scan[blockDim.x - 1];
        __syncthreads();
    }

    // Remaining elements handled similarly...
    uint32_t remainingElements = numBlocks - fullBlocksEnd;
    // reduction += s_scan[remainingElements - 1];

    if(tidx < numBlocks) {
        s_scan[tid] = passHist[tid + digitOffset];
    }

    s_scan[tid] = InclusiveWarpScan(s_scan[tid]);
    __syncthreads();

    if(tid < (blockDim.x >> LANE_LOG)) {
        s_scan[((tid + 1) << LANE_LOG) - 1] = ActiveInclusiveWarpScan(s_scan[((tid + 1) << LANE_LOG) - 1]);
    }
    __syncthreads();

    const uint32_t writeIdx = circularLaneShift + (tidx & ~LANE_MASK);
    if (writeIdx < numBlocks) {
        passHist[writeIdx + digitOffset] =
            (getLaneId() != LANE_MASK ? s_scan[tid] : 0) +
            (tid >= WARP_SIZE ?
            s_scan[(tid & ~LANE_MASK) - 1] : 0) +
            reduction;
    }

    // if (fullBlocksEnd + tid < numBlocks) {
    //     // Load remaining data with circular shift pattern
    //     s_scan[tid] = passHist[fullBlocksEnd + tid + digitOffset];    
    //     s_scan[tid] = InclusiveWarpScan(s_scan[tid]);
    //     __syncthreads();
   
    //     if (tid < blockDim.x / WARP_SIZE) {
    //         s_scan[((tid + 1) << LANE_LOG) - 1] = ActiveInclusiveWarpScan(s_scan[((tid + 1) << LANE_LOG) - 1]);
    //     }
    //     __syncthreads();

    //     // const uint32_t outputIdx = tid + (fullBlocksEnd & ~LANE_MASK);
    //     const uint32_t writeIdx = circularLaneShift + (fullBlocksEnd & ~LANE_MASK);
    //     if (writeIdx < numBlocks) {
    //         passHist[writeIdx + digitOffset] =
    //             (laneId != LANE_MASK ? s_scan[tid] : 0) +
    //             (tid >= WARP_SIZE ? 
    //                 s_scan[(tid & ~LANE_MASK) - 1] : 0) +
    //             reduction;
    //     }
    // }
    // __syncthreads();
    if(blockIdx.x == 1 && tid == numBlocks - 1) {
        printf("\nDigit[%u]\n", blockIdx.x);
        for(uint32_t i=0; i<blockDim.x; ++i) {
            printf("[%u %u] ", s_scan[i], i < numBlocks ? passHist[digitOffset + i] : 0);
        }
        printf("\n");
    }
}

template<typename T>
__global__ void RadixDownsweep(
    T* sort,              // Input array
    T* alt,               // Output array
    uint32_t* globalHist, // Global histogram
    uint32_t* passHist,   // Pass histogram
    uint32_t size,        // Total elements to sort
    uint32_t radixShift)  // Current radix shift amount
{
    // Shared memory layout
    // __shared__ uint32_t s_warpHistograms[N_THREADS * BIN_KEYS_PER_THREAD];  // blockDim.x * BIN_KEYS_PER_THREAD
    // __shared__ uint32_t s_localHistogram[RADIX];   // RADIX
    // volatile uint32_t* s_warpHist = &s_warpHistograms[WARP_INDEX << RADIX_LOG];

    // //clear shared memory
    // for (uint32_t i = threadIdx.x; i < BIN_HISTS_SIZE; i += blockDim.x)
    //     s_warpHistograms[i] = 0;

    // // Calculate thread's global position and valid key count
    // uint32_t thread_start = blockIdx.x * blockDim.x * BIN_KEYS_PER_THREAD + threadIdx.x;
    // uint32_t valid_keys = min(BIN_KEYS_PER_THREAD, 
    //                          (size - thread_start + WARP_SIZE - 1) / WARP_SIZE);
    
    // // Get ballot mask for partially filled warps
    // uint32_t ballot_mask = getBallotMask(thread_start, size);

    // // Clear shared memory
    // for (uint32_t i = threadIdx.x; i < BIN_HISTS_SIZE; i += blockDim.x) {
    //     s_warpHistograms[i] = 0;
    // }
    // __syncthreads();

    // // Load and convert keys
    // uint32_t keys[BIN_KEYS_PER_THREAD];
    // #pragma unroll
    // for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
    //     uint32_t idx = thread_start + i * WARP_SIZE;
    //     keys[i] = (idx < size && i < valid_keys) ? toBits<T>(sort[idx]) : 0;
    // }
    // __syncthreads();

    // // WLMS - Warp Level Multi Split
    // uint32_t offsets[BIN_KEYS_PER_THREAD];
    // #pragma unroll
    // for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
    //     unsigned warpFlags = ballot_mask;
        
    //     if (i < valid_keys) {
    //         #pragma unroll
    //         for (int k = 0; k < RADIX_LOG; ++k) {
    //             const bool t2 = keys[i] >> (k + radixShift) & 1;
    //             warpFlags &= (t2 ? 0 : ballot_mask) ^ __ballot_sync(ballot_mask, t2);
    //         }
    //     }

    //     const uint32_t bits = __popc(warpFlags & getLaneMaskLt());
    //     uint32_t preIncrementVal = 0;

    //     // Only the first thread in each digit group updates the histogram
    //     if (bits == 0 && i < valid_keys) {
    //         uint32_t digit = keys[i] >> radixShift & RADIX_MASK;
    //         preIncrementVal = atomicAdd(&s_warpHist[digit], __popc(warpFlags));
    //     }

    //     // Share the offset with other threads in the same digit group
    //     offsets[i] = __shfl_sync(ballot_mask, preIncrementVal, __ffs(warpFlags) - 1) + bits;
        
    //     // Validate offset is within bounds
    //     assert(offsets[i] < BIN_PART_SIZE);
    // }
    // __syncthreads();

    // // Exclusive prefix sum up the warp histograms
    // if (threadIdx.x < RADIX) {
    //     uint32_t reduction = s_warpHistograms[threadIdx.x];
    //     for (uint32_t i = threadIdx.x + RADIX; i < BIN_HISTS_SIZE; i += RADIX) {
    //         reduction += s_warpHistograms[i];
    //         s_warpHistograms[i] = reduction - s_warpHistograms[i];
    //     }

    //     s_warpHistograms[threadIdx.x] = InclusiveWarpScanCircularShift(reduction);
    // }
    // __syncthreads();

    // // Additional prefix sum processing
    // if (threadIdx.x < (RADIX >> WARP_LOG)) {
    //     s_warpHistograms[threadIdx.x << WARP_LOG] = 
    //         ActiveExclusiveWarpScan(s_warpHistograms[threadIdx.x << WARP_LOG]);
    // }
    // __syncthreads();

    // // Update offsets based on warp index
    // if (WARP_INDEX) {
    //     #pragma unroll 
    //     for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
    //         if (i < valid_keys) {
    //             const uint32_t digit = keys[i] >> radixShift & RADIX_MASK;
    //             offsets[i] += s_warpHist[digit] + s_warpHistograms[digit];
    //         }
    //     }
    // }
    // else {
    //     #pragma unroll
    //     for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
    //         if (i < valid_keys) {
    //             offsets[i] += s_warpHistograms[keys[i] >> radixShift & RADIX_MASK];
    //         }
    //     }
    // }

    // // Load block-level histogram data
    // if (threadIdx.x < RADIX) {
    //     s_localHistogram[threadIdx.x] = globalHist[threadIdx.x + (radixShift << 5)] +
    //         passHist[threadIdx.x * gridDim.x + blockIdx.x] - s_warpHistograms[threadIdx.x];
    // }
    // __syncthreads();

    // // Scatter keys to shared memory
    // #pragma unroll
    // for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
    //     if (i < valid_keys) {
    //         s_warpHistograms[offsets[i]] = keys[i];
    //     }
    // }
    // __syncthreads();

    // // Final scatter to global memory
    // uint32_t block_items = min(blockDim.x * BIN_KEYS_PER_THREAD, 
    //                           size - blockIdx.x * blockDim.x * BIN_KEYS_PER_THREAD);
    
    // for (uint32_t i = threadIdx.x; i < block_items; i += blockDim.x) {
    //     uint32_t digit = s_warpHistograms[i] >> radixShift & RADIX_MASK;
    //     uint32_t global_idx = s_localHistogram[digit] + i;
    //     if (global_idx < size) {
    //         // alt[global_idx] = fromBits<T>(s_warpHistograms[i]);
    //     }
    // }
}