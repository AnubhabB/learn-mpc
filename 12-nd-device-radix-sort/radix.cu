
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#define N_THREADS           256
#define RADIX               256             //Number of digit bins
#define WARP_SIZE           32
#define LANE_LOG            5               // LANE_LOG = 5 since 2^5 = 32 = warp size
#define RADIX_LOG           8

#define LANE_MASK           (WARP_SIZE - 1)
#define RADIX_MASK          (RADIX - 1)     //Mask of digit bins, to extract digits
#define WARP_INDEX          (threadIdx.x >> LANE_LOG)

#define BIN_KEYS_PER_THREAD 15
#define SUB_PARTITION_SIZE (BIN_KEYS_PER_THREAD * WARP_SIZE);

__device__ __forceinline__ uint32_t getLaneId() {
    uint32_t laneId;
    asm("mov.u32 %0, %%laneid;" : "=r"(laneId));
    return laneId;
}

// Prefix sum
// Performs an inclusive scan operation within a single warp
// Uses butterfly/sequential addressing pattern for efficiency
__device__ __forceinline__ uint32_t InclusiveWarpScan(uint32_t val) {
    #pragma unroll
    for (int offset = 1; offset <= WARP_SIZE; offset <<= 1) {
        const uint32_t t = __shfl_up_sync(0xffffffff, val, offset);
        if (getLaneId() >= offset) val += t;
    }

    return val;
}

// Circular shift prefix sum
__device__ __forceinline__ uint32_t InclusiveWarpScanCircularShift(uint32_t val) {
    #pragma unroll
    for (int offset = 1; offset <= WARP_SIZE; offset <<= 1) {
        const uint32_t t = __shfl_up_sync(0xffffffff, val, offset);
        if (getLaneId() >= offset) val += t;
    }

    return __shfl_sync(0xffffffff, val, getLaneId() + LANE_MASK & LANE_MASK);
}

// Helper for active warp scan (used for inter-warp scan) with early termination
__device__ __forceinline__ uint32_t ActiveInclusiveWarpScan(uint32_t val) {
    const uint32_t mask = __activemask();
    const int active_threads = __popc(mask);

    #pragma unroll
    for (int offset = 1; offset <= WARP_SIZE; offset <<= 1) {
        if (offset >= active_threads) break;  // Early termination
        const uint32_t t = __shfl_up_sync(mask, val, offset);
        if (getLaneId() >= offset) val += t;
    }

    return val;
}

// Prefix sum of active threads ot including itself
__device__ __forceinline__ uint32_t ActiveExclusiveWarpScan(uint32_t val) {
    const uint32_t mask = __activemask();
    #pragma unroll
    for (int offset = 1; offset <= WARP_SIZE; offset <<= 1) {
        const uint32_t t = __shfl_up_sync(mask, val, offset);
        if (getLaneId() >= offset) val += t;
    }

    const uint32_t t = __shfl_up_sync(mask, val, 1);
    return getLaneId() ? t : 0;
}

// Helper functions for bit conversions
template<typename T>
__device__ inline uint32_t toBits(T val) {
    if constexpr (std::is_same<T, float>::value) {
        return __float_as_uint(val) ^ ((__float_as_uint(val) >> 31) | 0x80000000);
    }
    else if constexpr (std::is_same<T, __half>::value) {
        uint16_t bits = __half_as_ushort(val);
        uint16_t mask = -int(bits >> 15) | 0x8000;
        return static_cast<uint32_t>(bits ^ mask);
    }
    else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
        uint16_t bits = __bfloat16_as_ushort(val);
        uint16_t mask = -int(bits >> 15) | 0x8000;
        return static_cast<uint32_t>(bits ^ mask);
    }
    else if constexpr (std::is_same<T, int64_t>::value) {
        // return static_cast<uint32_t>((val >> radixShift) & 0xFFFFFFFF);
        // TODO - how to handle int64?????
    }
    else {
        return static_cast<uint32_t>(val);
    }
}

// Vector type mappings
template<typename T> struct VectorTrait {
    // Default case (32-bit types)
    static constexpr uint32_t vector_size = 4;  // 4 * 4 bytes = 16 bytes
    static constexpr uint32_t bytes_per_vector = sizeof(T) * vector_size;
};

// Vectorizations for different types
template<>
struct VectorTrait<half> {
    static constexpr uint32_t vector_size = 8;  // 8 * 2 bytes = 16 bytes
    // using type = float4;
};

template<>
struct VectorTrait<hip_bfloat16> {
    static constexpr uint32_t vector_size = 8;   // 8 * 2 bytes = 16 bytes
};

// Specialization for 8-bit types
template<>
struct VectorTrait<uint8_t> {
    static constexpr uint32_t vector_size = 16;  // 16 * 1 byte = 16 bytes
};

// Vector type definitions
template<typename T>
struct alignas(16) Vector {  // Align to 16 bytes for optimal memory access
    T data[VectorTrait<T>::vector_size];
    
    __device__ __host__ T& operator[](int i) { return data[i]; }
    __device__ __host__ const T& operator[](int i) const { return data[i]; }
};

// Helper function to get type-specific maximum value
template<typename T>
__device__ inline T getTypeMax() {
    if constexpr (std::is_same<T, float>::value) {
        return INFINITY;
    }
    else if constexpr (std::is_same<T, __half>::value) {
        return __float2half(INFINITY);
    }
    else if constexpr (std::is_same<T, __hip_bfloat16>::value) {
        return __float2bfloat16(INFINITY);
    }
    else if constexpr (std::is_same<T, int64_t>::value) {
        return 0x7FFFFFFFFFFFFFFF;
    } else if constexpr (std::is_same<T, unsigned char>::value) {
        return 0xFF; // 255 in hex
    } else if constexpr (std::is_same<T, u_int32_t>::value) {
        return 0xFFFFFFFF;  // 4294967295 in hex
    } else {
        // This seems to be experimental
        // calling a constexpr __host__ function("max") from a __device__ function("getTypeMax") is not allowed. The experimental flag '--expt-relaxed-constexpr' can be used to allow this.
        
        // Shouldn't reach here
        return static_cast<T>(-1);
    }
}


template<typename T>
__global__ void RadixUpsweep(
    T* sort,
    uint32_t* globalHist,
    uint32_t* passHist,
    const uint32_t size,
    const uint32_t radixShift,
    const uint32_t partSize, // max number of elements being processed by this block
    const uint32_t vecPartSize // max number of `vector elements` in block
) {
    // Shared memory for histogram - two sections to avoid bank conflicts
    constexpr uint32_t sharedSize = RADIX * 2;
    __shared__ uint32_t s_globalHist[sharedSize];

    // Clear shared memory histogram
    #pragma unroll
    for (uint32_t i = threadIdx.x; i < sharedSize; i += blockDim.x)
        s_globalHist[i] = 0;
    __syncthreads();

    // Calculate this block's range
    const uint32_t block_start = blockIdx.x * partSize;
    const uint32_t block_end = min(block_start + partSize, size);
    const uint32_t elements_in_block = block_end - block_start;

    // Vector load based on types
    // constexpr uint32_t typesize = sizeof(T);
    using VecT = Vector<T>;
    constexpr uint32_t vec_size = VectorTrait<T>::vector_size;

    // Calculate number of full vectors - we are going to make an attempt to process 4 vectors at a time
    const uint32_t full_vecs = elements_in_block / vec_size;

    for (uint32_t i = threadIdx.x; i < full_vecs; i += blockDim.x) {
        const uint32_t idx = block_start / vec_size + i;
        const VecT vec_val = reinterpret_cast<const VecT*>(sort)[idx];
        
        #pragma unroll
        for (int j = 0; j < vec_size; j++) {
            uint32_t bits = toBits(vec_val[j]);
            atomicAdd(&s_globalHist[bits >> radixShift & RADIX_MASK], 1);
        }
    }

    // Process remaining elements
    const uint32_t vec_end = block_start + (full_vecs * vec_size);
    for (uint32_t i = threadIdx.x + vec_end; i < block_end; i += blockDim.x) {
        const T t = sort[i];
        uint32_t bits = toBits(t);
        atomicAdd(&s_globalHist[bits >> radixShift & RADIX_MASK], 1);
    }

    __syncthreads();

    // Reduce histograms and prepare for prefix sum
    for (uint32_t i = threadIdx.x; i < RADIX; i += blockDim.x) {
        s_globalHist[i] += s_globalHist[i + RADIX];
        passHist[i * gridDim.x + blockIdx.x] = s_globalHist[i];
        s_globalHist[i] = InclusiveWarpScanCircularShift(s_globalHist[i]);
    }   
    __syncthreads();

    // Perform warp-level scan
    if (threadIdx.x < (RADIX >> LANE_LOG))
        s_globalHist[threadIdx.x << LANE_LOG] = ActiveExclusiveWarpScan(s_globalHist[threadIdx.x << LANE_LOG]);
    __syncthreads();

    // Update global histogram with prefix sum results
    for (uint32_t i = threadIdx.x; i < RADIX; i += blockDim.x) {
        atomicAdd(
            &globalHist[i + (radixShift << LANE_LOG)], 
            s_globalHist[i] + 
            (getLaneId() ? 
                __shfl_sync(0xfffffffe, s_globalHist[i - 1], 1) : 0)
        );
    }
}


// TODO: optimize this with shared memory
__global__ void RadixScan(
    uint32_t* passHist,
    const uint32_t threadBlocks
) {
    const uint32_t lane_id = getLaneId();
    const uint32_t digit_offset = blockIdx.x * threadBlocks;
    
    // Process in chunks of WARP_SIZE
    const uint32_t num_warps = (threadBlocks + WARP_SIZE - 1) / WARP_SIZE;
    uint32_t running_sum = 0;
    
    // Process each warp-sized chunk
    for (uint32_t warp = 0; warp < num_warps; warp++) {
        const uint32_t start_idx = warp * WARP_SIZE;
        const uint32_t local_idx = start_idx + lane_id;
        
        // Load and scan within warp
        uint32_t val = 0;
        if (local_idx < threadBlocks) {
            val = passHist[digit_offset + local_idx];
        }
        
        // Perform inclusive scan within warp
        val = InclusiveWarpScan(val);
        
        // Add running sum from previous iterations
        val += running_sum;
        
        // Store result if within bounds
        if (local_idx < threadBlocks) {
            passHist[digit_offset + local_idx] = val;
        }
        
        // Update running sum for next iteration
        // Get the last valid value in this warp
        uint32_t warp_last = __shfl_sync(0xffffffff, val, min(threadBlocks - start_idx, WARP_SIZE) - 1);
        if (lane_id == 0) {
            running_sum = warp_last;
        }
    }
}

template<typename T>
__global__ void RadixDownsweep(
    T* sort,
    T* sortAlt,
    uint32_t* payload,
    uint32_t* payloadAlt,
    uint32_t* globalHist,
    uint32_t* passHist,
    const uint32_t size,
    const uint32_t radixShift
) {
    constexpr uint32_t s_histSize   = RADIX * (N_THREADS / WARP_SIZE);
    constexpr uint32_t elemPerBlock = N_THREADS * BIN_KEYS_PER_THREAD;
    constexpr uint32_t elemPerWarp  = WARP_SIZE * BIN_KEYS_PER_THREAD;
    
    // Shared memory histogram
    __shared__ uint32_t s_warpHistograms[s_histSize];
    __shared__ uint32_t s_localHistogram[RADIX];

    // Each warp's histogram section
    volatile uint32_t* s_warpHist = &s_warpHistograms[WARP_INDEX << RADIX_LOG];

    //clear shared memory
    for (uint32_t i = threadIdx.x; i < s_histSize; i += blockDim.x)
        s_warpHistograms[i] = 0;

    //load keys
    T keys[BIN_KEYS_PER_THREAD];

    // Calculate base index for this block
    uint32_t blockStart = blockIdx.x * elemPerBlock;
    // Calculate base index for this warp within the block
    uint32_t warpStart = blockStart + (WARP_INDEX * elemPerWarp);
    // Calculate thread's starting position within warp
    uint32_t threadStart = warpStart + getLaneId();
    
    if (blockStart + elemPerBlock <= size) {
        // Full block case - no bounds checking needed
        #pragma unroll
        for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
            keys[i] = sort[threadStart + i * WARP_SIZE];
        }
    } else {
        // Last block case - needs bounds checking
        #pragma unroll
        for (uint32_t i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
            uint32_t loadIndex = threadStart + i * WARP_SIZE;
            keys[i] = loadIndex < size ? sort[loadIndex] : getTypeMax<T>();
        }
    }

    __syncthreads();
}